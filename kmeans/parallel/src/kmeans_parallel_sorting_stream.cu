#include "hip/hip_runtime.h"
#include "kmeans_parallel.cuh"
#include "announce.hh"
#include <algorithm>
#include "log.cc"

static const int UpdateCentroidBlockDim = 1024;

void sortAndGetLabelCounts(DataPoint* const, size_t* const, size_t* const, size_t* const);
void memcpyCentroidsToConst(DataPoint*);
void memcpyCentroidsFromConst(DataPoint*);
void memcpyLabelCountToConst(size_t*, size_t*, size_t*);
void memcpyLabelCountFromConst(size_t*, size_t*, size_t*);

__device__ __constant__ Data_T constCentroidValues[KSize*FeatSize];
__device__ __constant__ size_t constLabelCounts[KSize];
__device__ __constant__ size_t constLabelFirstIdxes[KSize];
__device__ __constant__ size_t constLabelLastIdxes[KSize];

#define Trans_DataValues_IDX(x,y) y*DataSize+x
#define CentroidValues_IDX(x,y) y*FeatSize+x
void transposeDataPointers(const DataPoint* const data, Labels_T labels, Trans_DataValues transposed);
void untransposeDataPointers(const Trans_DataValues transposed, Labels_T labels, DataPoint* const data);

void KMeans::main(DataPoint* const centroids, DataPoint* const data) {
    Log<> log("./results/parallel");

    Labels_T dataLabels = new Label_T[DataSize];
    Trans_DataValues dataValuesTransposed = new Data_T[FeatSize * DataSize];

    transposeDataPointers(data, dataLabels, dataValuesTransposed);

    cudaAssert (
        hipHostRegister(dataLabels, DataSize*sizeof(Label_T), hipHostRegisterPortable)
    );
    cudaAssert (
        hipHostRegister(dataValuesTransposed, FeatSize*DataSize*sizeof(Data_T), hipHostRegisterPortable)
    );
    cudaAssert (
        hipHostRegister(centroids, KSize*sizeof(DataPoint), hipHostRegisterPortable)
    );

    auto labelCounts = new size_t[KSize]{0,};
    auto labelFirstIdxes = new size_t[KSize]{0,};
    auto labelLastIdxes = new size_t[KSize]{0,};
    auto newCentroids = new DataPoint[KSize];
    auto isUpdated = new bool(true);

    cudaAssert (
        hipHostRegister(newCentroids, KSize*sizeof(DataPoint), hipHostRegisterPortable)
    );
    cudaAssert (
        hipHostRegister(isUpdated, sizeof(bool), hipHostRegisterPortable)
    );

    int numThread_labeling = 256; /*TODO get from study*/
    int numBlock_labeling = ceil((float)DataSize / numThread_labeling);

    int threashold = 5;
    while(threashold-- > 0) {
        hipDeviceSynchronize();
        memcpyCentroidsToConst(centroids);
        KMeans::labeling<<<numBlock_labeling, numThread_labeling>>>(DataSize, FeatSize, dataLabels, dataValuesTransposed);

        hipDeviceSynchronize();
        untransposeDataPointers(dataValuesTransposed, dataLabels, data);
        sortAndGetLabelCounts(data, labelCounts, labelFirstIdxes, labelLastIdxes);
        transposeDataPointers(data, dataLabels, dataValuesTransposed);
        announce.Labels(data);

        memcpyLabelCountToConst(labelCounts, labelFirstIdxes, labelLastIdxes);
        size_t maxLabelCount = 0;
        for(int i=0; i!=KSize; ++i)
            maxLabelCount = std::max(maxLabelCount, labelCounts[i]);

        resetNewCentroids<<<KSize,FeatSize>>>(newCentroids);

        dim3 dimBlock(UpdateCentroidBlockDim, 1, 1);
        dim3 dimGrid(ceil(maxLabelCount/UpdateCentroidBlockDim), KSize, 1);
        KMeans::updateCentroidAccum<<<dimGrid, dimBlock>>>(newCentroids, dataValuesTransposed);
        KMeans::updateCentroidDivide<<<KSize, FeatSize>>>(newCentroids);

        KMeans::checkIsSame<<<KSize, FeatSize>>>(isUpdated, centroids, newCentroids);
        hipDeviceSynchronize();
        if(*isUpdated)
            break;
        *isUpdated = true;

        memcpyCentroid<<<KSize,FeatSize>>>(centroids, newCentroids);
    }

    hipDeviceSynchronize();
    cudaAssert( hipPeekAtLastError());
    announce.Labels(data);
    announce.InitCentroids(newCentroids);

    cudaAssert( hipHostUnregister(dataLabels));
    cudaAssert( hipHostUnregister(dataValuesTransposed));
    cudaAssert( hipHostUnregister(centroids) );
    cudaAssert( hipHostUnregister(newCentroids) );
    cudaAssert( hipHostUnregister(isUpdated) );

    delete[] dataLabels;
    delete[] dataValuesTransposed;
    delete[] labelCounts;
    delete[] labelFirstIdxes;
    delete[] labelLastIdxes;
    delete[] newCentroids;
    delete isUpdated;
    log.Lap("KMeans-Parallel End");
}

__global__
void KMeans::labeling(size_t dataSize, size_t featSize, Labels_T const labels, Trans_DataValues const data) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= DataSize)
        return;

    Data_T distSQRSums[KSize]{0,};

    for(int i=0; i!=FeatSize; ++i) {
        Data_T currValue = data[i*dataSize + idx];

        for(int j=0; j!=KSize; ++j) {
            Data_T currDist = currValue - constCentroidValues[j*featSize + i];
            distSQRSums[j] += currDist * currDist;
        }
	__syncthreads();
    }

    Data_T minDistSQRSum = MaxDataValue;
    Label_T minDistLabel = 0;
    for(int i=0; i!=KSize; ++i) {
        if(minDistSQRSum > distSQRSums[i]) {
            minDistSQRSum = distSQRSums[i];
            minDistLabel = i;
        }
    }

    labels[idx] = minDistLabel;
}

/// update centroids //////////////////////////////////////////////////////////////////////////////
// blockDim = 8~32 ~ 128
// gridDim = 10, ceil(maxLabelCount/blockDim)
// width = maxLabelCount
// blockIdx.y = label
__global__
void KMeans::updateCentroidAccum(DataPoint* const centroids, const Trans_DataValues data) {
    __shared__ Data_T Sum[UpdateCentroidBlockDim];

    const int tID = threadIdx.x;
    const Label_T label = blockIdx.y;

    const size_t labelFirstIdx = constLabelFirstIdxes[label];
    const size_t labelLastIdx = constLabelLastIdxes[label];
    const size_t dataIdx = labelFirstIdx + (blockIdx.x * blockDim.x + tID);

    if(dataIdx > labelLastIdx)
        return;

    {//\Asserts
    assert(label >= 0 && label < 10);
    assert(labelLastIdx < DataSize);
    assert(dataIdx >= labelFirstIdx);
    }

    for(int featIdx=0; featIdx!=FeatSize; ++featIdx) {
        Sum[tID] = data[Trans_DataValues_IDX(dataIdx, featIdx)];
        __syncthreads();// TODO 없어도 되나?

        for(int stride=blockDim.x/2; stride>=1; stride>>=1) {
            if(tID < stride && dataIdx+stride <= labelLastIdx)
                Sum[tID] += Sum[tID+stride];
            __syncthreads();
        }

        if(tID != 0)
            continue;

        atomicAdd(&(centroids[label].value[featIdx]), Sum[tID]);
    }
}

__global__
void KMeans::updateCentroidDivide(DataPoint* const centroids) {
    int label = blockIdx.x;
    centroids[label].value[threadIdx.x] /= constLabelCounts[label];
}

void sortAndGetLabelCounts (
    DataPoint* const data,
    size_t* const labelCounts,
    size_t* const labelFirstIdxes,
    size_t* const labelLastIdxes
) {
    std::sort(data, data+DataSize, cmpDataPoint);

    const DataPoint* dataPtr = data;

    Label_T currLabel = dataPtr->label;
    int currLabelCount = 0;
    labelFirstIdxes[currLabelCount] = 0;

    for(int i=0; i!=DataSize; ++i) {
        if(currLabel != dataPtr->label) {
            labelCounts[currLabel] = currLabelCount;
            labelLastIdxes[currLabel] = i-1;

            currLabelCount = 0;
            currLabel = dataPtr->label;
            labelFirstIdxes[currLabel] = i;
        }
        currLabelCount++;
        dataPtr++;
    }
    labelCounts[currLabel] = currLabelCount;
    labelLastIdxes[currLabel] = DataSize-1;
}

void memcpyCentroidsToConst(DataPoint* centroids) {
    Data_T values[KSize*FeatSize];
    
    for(int i=0; i!=KSize; ++i) {
        for(int j=0; j!=FeatSize; ++j) {
            values[i*FeatSize+j] = centroids[i].value[j];
        }
    }
    cudaAssert (
        hipMemcpyToSymbol(HIP_SYMBOL(constCentroidValues), values, KSize*FeatSize*sizeof(Data_T))
    );
}

void memcpyCentroidsFromConst(DataPoint* centroids) {
    Label_T labels[KSize];
    Data_T values[KSize*FeatSize];

    cudaAssert (
        hipMemcpyFromSymbol(values, HIP_SYMBOL(constCentroidValues), KSize*FeatSize*sizeof(Data_T))
    );

    for(int i=0; i!=KSize; ++i) {
        centroids[i].label = labels[i];

        for(int j=0; j!=FeatSize; ++j) {
            centroids[i].value[j] = values[i*FeatSize+j];
        }
    }
}

void memcpyLabelCountToConst(size_t* labelCount, size_t* labelFirstIdxes, size_t* labelLastIdxes) {
    cudaAssert (
        hipMemcpyToSymbol(HIP_SYMBOL(constLabelCounts), labelCount, KSize*sizeof(size_t))
    );
    cudaAssert (
        hipMemcpyToSymbol(HIP_SYMBOL(constLabelFirstIdxes), labelFirstIdxes, KSize*sizeof(size_t))
    );
    cudaAssert (
        hipMemcpyToSymbol(HIP_SYMBOL(constLabelLastIdxes), labelLastIdxes, KSize*sizeof(size_t))
    );
}

void memcpyLabelCountFromConst(size_t* labelCount, size_t* labelFirstIdxes, size_t* labelLastIdxes) {
    cudaAssert (
        hipMemcpyFromSymbol(labelCount, HIP_SYMBOL(constLabelCounts), KSize*sizeof(size_t))
    );
    cudaAssert (
        hipMemcpyFromSymbol(labelFirstIdxes, HIP_SYMBOL(constLabelFirstIdxes), KSize*sizeof(size_t))
    );
    cudaAssert (
        hipMemcpyFromSymbol(labelLastIdxes, HIP_SYMBOL(constLabelLastIdxes), KSize*sizeof(size_t))
    );
}

void transposeDataPointers(const DataPoint* const data, Labels_T labels, Trans_DataValues transposed) {
    for(int i=0; i!=DataSize; ++i) {
        labels[i] = data[i].label;

        for(int j=0; j!=FeatSize; ++j) {
            transposed[j*DataSize + i] = data[i].value[j];
        }
    }
}

void untransposeDataPointers(const Trans_DataValues transposed, Labels_T labels, DataPoint* const data) {
    for(int i=0; i!=DataSize; ++i) {
        data[i].label = labels[i];

        for(int j=0; j!=FeatSize; ++j) {
            data[i].value[j] = transposed[j*DataSize + i];
        }
    }
}
