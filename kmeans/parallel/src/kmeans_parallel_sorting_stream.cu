#include "hip/hip_runtime.h"
#include "kmeans_parallel.cuh"
#include "announce.hh"
#include <algorithm>
//#include <hip/hiprtc.h>

static const int UpdateCentroidBlockDim= 32;

void sortAndGetLabelCounts(DataPoint* const, size_t* const, size_t* const, size_t* const);
void memcpyCentroidsToConst(DataPoint*);
void memcpyCentroidsFromConst(DataPoint*);
void memcpyLabelCountToConst(size_t*, size_t*, size_t*);
void memcpyLabelCountFromConst(size_t*, size_t*, size_t*);

__device__ __constant__ Data_T constCentroidValues[KSize*FeatSize];
__device__ __constant__ size_t constLabelCounts[KSize];
__device__ __constant__ size_t constLabelFirstIdxes[KSize];
__device__ __constant__ size_t constLabelLastIdxes[KSize];


void KMeans::main(DataPoint* const centroids, DataPoint* const data) {
    cudaAssert (
        hipHostRegister(data, DataSize*sizeof(DataPoint), hipHostRegisterPortable)
    );
    cudaAssert (
        hipHostRegister(centroids, KSize*sizeof(DataPoint), hipHostRegisterPortable)
    );

    auto labelCounts = new size_t[KSize]{0,};
    auto labelFirstIdxes = new size_t[KSize]{0,};
    auto labelLastIdxes = new size_t[KSize]{0,};
    auto newCentroids = new DataPoint[KSize];
    bool* isSame = new bool(true);

    cudaAssert (
        hipHostRegister(newCentroids, KSize*sizeof(DataPoint), hipHostRegisterPortable)
    );
    cudaAssert (
        hipHostRegister(isSame, sizeof(bool), hipHostRegisterPortable)
    );

    //study(deviceQuery());
    int numThread_labeling = 8; /*TODO get from study*/
    int numBlock_labeling = ceil((float)DataSize / numThread_labeling);

    int threashold = 2;
    while(threashold-- > 0) {
        hipDeviceSynchronize();
        memcpyCentroidsToConst(centroids);
        KMeans::labeling<<<numBlock_labeling, numThread_labeling>>>(data);

        hipDeviceSynchronize();
        sortAndGetLabelCounts(data, labelCounts, labelFirstIdxes, labelLastIdxes);
        memcpyLabelCountToConst(labelCounts, labelFirstIdxes, labelLastIdxes);
        size_t maxLabelCount = 0;
        for(int i=0; i!=KSize; ++i)
            maxLabelCount = std::max(maxLabelCount, labelCounts[i]);

        resetNewCentroids<<<KSize,FeatSize>>>(newCentroids);

        dim3 dimBlock(UpdateCentroidBlockDim, 1, 1);
        dim3 dimGrid(ceil(maxLabelCount/UpdateCentroidBlockDim), KSize, 1);
        KMeans::updateCentroidAccum<<<dimGrid, dimBlock>>>(newCentroids, data);
        KMeans::updateCentroidDivide<<<KSize, FeatSize>>>(newCentroids);

        KMeans::checkIsSame<<<KSize, FeatSize>>>(isSame, centroids, newCentroids);
        //hipDeviceSynchronize();
        //if(isSame)
            //break;

        memcpyCentroid<<<KSize,FeatSize>>>(centroids, newCentroids);
    }

    hipDeviceSynchronize();
    cudaAssert( hipPeekAtLastError());
    announce.Labels(data);
    announce.InitCentroids(newCentroids);

    cudaAssert( hipHostUnregister(data) );
    cudaAssert( hipHostUnregister(centroids) );
    cudaAssert( hipHostUnregister(newCentroids) );
    cudaAssert( hipHostUnregister(isSame) );

    delete[] labelCounts;
    delete[] labelFirstIdxes;
    delete[] labelLastIdxes;
    delete[] newCentroids;
    delete isSame;
}

/// labeling ////////////////////////////////////////////////////////////////////////////////////
__global__
void KMeans::labeling(DataPoint* const data) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= DataSize)
        return;

    DataPoint threadData = data[idx];

    Label_T minDistLabel = 0;
    Data_T minDistSQR = MaxDataValue;

    for(int i=0; i!=KSize; ++i) {
        Data_T currDistSQR = KMeans::Labeling::euclideanDistSQR(threadData.value, constCentroidValues + i*FeatSize);
        if(minDistSQR > currDistSQR) {
            minDistLabel = i;
            minDistSQR = currDistSQR;
        }
    }

    data[idx].label = minDistLabel;
}

__device__ 
Data_T KMeans::Labeling::euclideanDistSQR ( const Data_T* const __restrict__ lhs, const Data_T* const __restrict__ rhs) { 
    const Data_T* valuePtrLHS = lhs;
    const Data_T* valuePtrRHS = rhs;

    Data_T distSQR = 0;

    for(int i=0; i!=FeatSize; ++i) {
        Data_T dist = *valuePtrLHS - *valuePtrRHS;

        distSQR += dist*dist;

        valuePtrLHS++;
        valuePtrRHS++;
    }

    return distSQR;
}

/// update centroids //////////////////////////////////////////////////////////////////////////////
// blockDim = 8~32 ~ 128
// gridDim = 10, ceil(maxLabelCount/blockDim)
// width = maxLabelCount
// blockIdx.y = label
__global__
void KMeans::updateCentroidAccum(DataPoint* const centroids, const DataPoint* const data) {
    __shared__ Data_T Sum[UpdateCentroidBlockDim][FeatSize];

    const int tID = threadIdx.x;
    const Label_T label = blockIdx.y;

    const size_t labelFirstIdx = constLabelFirstIdxes[label]; // TODO Const mem으로 보내기
    const size_t labelLastIdx = constLabelLastIdxes[label]; // TODO Const mem으로 보내기
    const size_t dataIdx = labelFirstIdx + (blockIdx.x * blockDim.x + tID);

    if(dataIdx > labelLastIdx)
        return;

    for(int i=0; i!=FeatSize; ++i)
        Sum[tID][i] = data[dataIdx].value[i];
    __syncthreads();// TODO 없어도 되나?

    {//\Asserts
    assert(label >= 0 && label < 10);
    assert(labelLastIdx < DataSize);
    assert(data[dataIdx].label == label);
    }

    for(int stride=blockDim.x/2; stride>=1; stride>>=1) {
        if(tID < stride && dataIdx+stride <= labelLastIdx)
            Update::addValuesLtoR(Sum[tID+stride], Sum[tID]);
        __syncthreads();
    }

    if(tID != 0)
        return;

    for(int i=0; i!=FeatSize; ++i)
        atomicAdd(&(centroids[label].value[i]), Sum[tID][i]);
}

__global__
void KMeans::updateCentroidDivide(DataPoint* const centroids) {
    int label = blockIdx.x;
    centroids[label].value[threadIdx.x] /= constLabelCounts[label];
}

__device__
void KMeans::Update::addValuesLtoR(const Data_T* const lhs, Data_T* const rhs) {
    const Data_T* lhsPtr = lhs;
    Data_T* rhsPtr = rhs;

    for(int i=0; i!=FeatSize; ++i)
        *(rhsPtr++) += *(lhsPtr++);
}

void sortAndGetLabelCounts (
    DataPoint* const data,
    size_t* const labelCounts,
    size_t* const labelFirstIdxes,
    size_t* const labelLastIdxes
) {
    std::sort(data, data+DataSize, cmpDataPoint);

    const DataPoint* dataPtr = data;

    Label_T currLabel = dataPtr->label;
    int currLabelCount = 0;
    labelFirstIdxes[currLabelCount] = 0;

    for(int i=0; i!=DataSize; ++i) {
        if(currLabel != dataPtr->label) {
            labelCounts[currLabel] = currLabelCount;
            labelLastIdxes[currLabel] = i-1;

            currLabelCount = 0;
            currLabel = dataPtr->label;
            labelFirstIdxes[currLabel] = i;
        }
        currLabelCount++;
        dataPtr++;
    }
    labelCounts[currLabel] = currLabelCount;
    labelLastIdxes[currLabel] = DataSize-1;
}

void memcpyCentroidsToConst(DataPoint* centroids) {
    Data_T values[KSize*FeatSize];
    
    for(int i=0; i!=KSize; ++i) {
        for(int j=0; j!=FeatSize; ++j) {
            values[i*FeatSize+j] = centroids[i].value[j];
        }
    }
    cudaAssert (
        hipMemcpyToSymbol(HIP_SYMBOL(constCentroidValues), values, KSize*FeatSize*sizeof(Data_T))
    );
}

void memcpyCentroidsFromConst(DataPoint* centroids) {
    Label_T labels[KSize];
    Data_T values[KSize*FeatSize];

    cudaAssert (
        hipMemcpyFromSymbol(values, HIP_SYMBOL(constCentroidValues), KSize*FeatSize*sizeof(Data_T))
    );

    for(int i=0; i!=KSize; ++i) {
        centroids[i].label = labels[i];

        for(int j=0; j!=FeatSize; ++j) {
            centroids[i].value[j] = values[i*FeatSize+j];
        }
    }
}

void memcpyLabelCountToConst(size_t* labelCount, size_t* labelFirstIdxes, size_t* labelLastIdxes) {
    cudaAssert (
        hipMemcpyToSymbol(HIP_SYMBOL(constLabelCounts), labelCount, KSize*sizeof(size_t))
    );
    cudaAssert (
        hipMemcpyToSymbol(HIP_SYMBOL(constLabelFirstIdxes), labelFirstIdxes, KSize*sizeof(size_t))
    );
    cudaAssert (
        hipMemcpyToSymbol(HIP_SYMBOL(constLabelLastIdxes), labelLastIdxes, KSize*sizeof(size_t))
    );
}

void memcpyLabelCountFromConst(size_t* labelCount, size_t* labelFirstIdxes, size_t* labelLastIdxes) {
    cudaAssert (
        hipMemcpyFromSymbol(labelCount, HIP_SYMBOL(constLabelCounts), KSize*sizeof(size_t))
    );
    cudaAssert (
        hipMemcpyFromSymbol(labelFirstIdxes, HIP_SYMBOL(constLabelFirstIdxes), KSize*sizeof(size_t))
    );
    cudaAssert (
        hipMemcpyFromSymbol(labelLastIdxes, HIP_SYMBOL(constLabelLastIdxes), KSize*sizeof(size_t))
    );
}