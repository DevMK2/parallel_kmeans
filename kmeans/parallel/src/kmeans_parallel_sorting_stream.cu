#include "hip/hip_runtime.h"
#include "kmeans_parallel.cuh"
#include "announce.hh"
#include <algorithm>
#include "log.cc"

static const int UpdateCentroidBlockDim = 256;

void sortAndGetLabelCounts(DataPoint* const, size_t* const, size_t* const, size_t* const);
void memcpyCentroidsToConst(DataPoint*);
void memcpyCentroidsFromConst(DataPoint*);
void memcpyLabelCountToConst(size_t*, size_t*, size_t*);
void memcpyLabelCountFromConst(size_t*, size_t*, size_t*);

__device__ __constant__ Data_T constCentroidValues[KSize*FeatSize];
__device__ __constant__ size_t constLabelCounts[KSize];
__device__ __constant__ size_t constLabelFirstIdxes[KSize];
__device__ __constant__ size_t constLabelLastIdxes[KSize];

#define Trans_DataValues_IDX(x,y) y*DataSize+x
#define CentroidValues_IDX(x,y) y*FeatSize+x

Labels_T dataLabels;
Trans_DataValues dataValuesTransposed;

void transposDataPointers(const DataPoint* const data, Trans_DataValues transposed) {
    for(int i=0; i!=DataSize; ++i) {
        dataLabels[i] = data[i].label;

        for(int j=0; j!=FeatSize; ++j) {
            transposed[j*DataSize + i] = data[i].value[j];
        }
    }

    // TODO Delete me
    for(int i=0; i!=FeatSize; ++i)
        for(int j=0; j!=DataSize; ++j)
            assert(transposed[i*DataSize + j] == data[j].value[i]);
}

void untransposDataPointers(const Trans_DataValues transposed, DataPoint* const data) {
    for(int i=0; i!=DataSize; ++i) {
        data[i].label = dataLabels[i];

        for(int j=0; j!=FeatSize; ++j) {
            data[i].value[j] = transposed[j*DataSize + i];
        }
    }
}

void KMeans::main(DataPoint* const centroids, DataPoint* const data) {
    Log<> log("./results/parallel");
    transposDataPointers(data, dataValuesTransposed);

    cudaAssert (
        hipHostRegister(dataLabels, DataSize*sizeof(Label_T), hipHostRegisterPortable)
    );
    cudaAssert (
        hipHostRegister(dataValuesTransposed, FeatSize*DataSize*sizeof(Data_T), hipHostRegisterPortable)
    );
    cudaAssert (
        hipHostRegister(centroids, KSize*sizeof(DataPoint), hipHostRegisterPortable)
    );

    auto labelCounts = new size_t[KSize]{0,};
    auto labelFirstIdxes = new size_t[KSize]{0,};
    auto labelLastIdxes = new size_t[KSize]{0,};
    auto newCentroids = new DataPoint[KSize];
    bool* isSame = new bool(true);

    cudaAssert (
        hipHostRegister(newCentroids, KSize*sizeof(DataPoint), hipHostRegisterPortable)
    );
    cudaAssert (
        hipHostRegister(isSame, sizeof(bool), hipHostRegisterPortable)
    );

    //study(deviceQuery());
    int numThread_labeling = 512; /*TODO get from study*/
    int numBlock_labeling = ceil((float)DataSize / numThread_labeling);

    int threashold = 20;
    while(threashold-- > 0) {
        hipDeviceSynchronize();
        memcpyCentroidsToConst(centroids);
        KMeans::labeling<<<numBlock_labeling, numThread_labeling>>>(&dataLabels, &dataValuesTransposed);

        hipDeviceSynchronize();
        untransposDataPointers(dataValuesTransposed, data);
        sortAndGetLabelCounts(data, labelCounts, labelFirstIdxes, labelLastIdxes);
        transposDataPointers(data, dataValuesTransposed);
        announce.Labels(data);

        memcpyLabelCountToConst(labelCounts, labelFirstIdxes, labelLastIdxes);
        size_t maxLabelCount = 0;
        for(int i=0; i!=KSize; ++i)
            maxLabelCount = std::max(maxLabelCount, labelCounts[i]);

        resetNewCentroids<<<KSize,FeatSize>>>(newCentroids);

        dim3 dimBlock(UpdateCentroidBlockDim, 1, 1);
        dim3 dimGrid(ceil(maxLabelCount/UpdateCentroidBlockDim), KSize, 1);
        KMeans::updateCentroidAccum<<<dimGrid, dimBlock>>>(newCentroids, dataValuesTransposed);
        KMeans::updateCentroidDivide<<<KSize, FeatSize>>>(newCentroids);

        //announce.InitCentroids(newCentroids);

        KMeans::checkIsSame<<<KSize, FeatSize>>>(isSame, centroids, newCentroids);
        hipDeviceSynchronize();
        if(*isSame)
            break;
        *isSame = true;

        memcpyCentroid<<<KSize,FeatSize>>>(centroids, newCentroids);
    }

    hipDeviceSynchronize();
    cudaAssert( hipPeekAtLastError());
    announce.Labels(data);
    announce.InitCentroids(newCentroids);

    cudaAssert( hipHostUnregister(dataLabels));
    cudaAssert( hipHostUnregister(dataValuesTransposed));
    cudaAssert( hipHostUnregister(centroids) );
    cudaAssert( hipHostUnregister(newCentroids) );
    cudaAssert( hipHostUnregister(isSame) );

    delete[] labelCounts;
    delete[] labelFirstIdxes;
    delete[] labelLastIdxes;
    delete[] newCentroids;
    delete isSame;
    log.Lap("KMeans-Parallel End");
}

/// labeling ////////////////////////////////////////////////////////////////////////////////////
__global__
void KMeans::labeling(Labels_T* const labels, Trans_DataValues* const data) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= DataSize)
        return;

    Data_T distSQRSums[KSize]{0,};

    for(int i=0; i!=FeatSize; ++i) {
        Data_T currValue = (*data)[Trans_DataValues_IDX(idx, i)];

        for(int j=0; j!=KSize; ++j) {
            Data_T currDist = currValue - constCentroidValues[CentroidValues_IDX(i,j)];
            distSQRSums[j] += currDist * currDist;
        }
    }

    Data_T minDistSQRSum = MaxDataValue;
    Label_T minDistLabel = 0;
    for(int i=0; i!=KSize; ++i) {
        if(minDistSQRSum > distSQRSums[i]) {
            minDistSQRSum = distSQRSums[i];
            minDistLabel = i;
        }
    }

    (*labels)[idx]= minDistLabel;
}

/// update centroids //////////////////////////////////////////////////////////////////////////////
// blockDim = 8~32 ~ 128
// gridDim = 10, ceil(maxLabelCount/blockDim)
// width = maxLabelCount
// blockIdx.y = label
__global__
void KMeans::updateCentroidAccum(DataPoint* const centroids, const Trans_DataValues data) {
    __shared__ Data_T Sum[UpdateCentroidBlockDim];

    const int tID = threadIdx.x;
    const Label_T label = blockIdx.y;

    const size_t labelFirstIdx = constLabelFirstIdxes[label];
    const size_t labelLastIdx = constLabelLastIdxes[label];
    const size_t dataIdx = labelFirstIdx + (blockIdx.x * blockDim.x + tID);

    if(dataIdx > labelLastIdx)
        return;

    {//\Asserts
    assert(label >= 0 && label < 10);
    assert(labelLastIdx < DataSize);
    assert(dataIdx >= labelFirstIdx);
    }

    for(int featIdx=0; featIdx!=FeatSize; ++featIdx) {
        Sum[tID] = data[Trans_DataValues_IDX(dataIdx, featIdx)];
        __syncthreads();// TODO 없어도 되나?

        for(int stride=blockDim.x/2; stride>=1; stride>>=1) {
            if(tID < stride && dataIdx+stride <= labelLastIdx)
                Sum[tID] += Sum[tID+stride];
            __syncthreads();
        }

        if(tID != 0)
            continue;

        atomicAdd(&(centroids[label].value[featIdx]), Sum[tID]);
    }
}

__global__
void KMeans::updateCentroidDivide(DataPoint* const centroids) {
    int label = blockIdx.x;
    centroids[label].value[threadIdx.x] /= constLabelCounts[label];
}

void sortAndGetLabelCounts (
    DataPoint* const data,
    size_t* const labelCounts,
    size_t* const labelFirstIdxes,
    size_t* const labelLastIdxes
) {
    std::sort(data, data+DataSize, cmpDataPoint);

    const DataPoint* dataPtr = data;

    Label_T currLabel = dataPtr->label;
    int currLabelCount = 0;
    labelFirstIdxes[currLabelCount] = 0;

    for(int i=0; i!=DataSize; ++i) {
        if(currLabel != dataPtr->label) {
            labelCounts[currLabel] = currLabelCount;
            labelLastIdxes[currLabel] = i-1;

            currLabelCount = 0;
            currLabel = dataPtr->label;
            labelFirstIdxes[currLabel] = i;
        }
        currLabelCount++;
        dataPtr++;
    }
    labelCounts[currLabel] = currLabelCount;
    labelLastIdxes[currLabel] = DataSize-1;
}

void memcpyCentroidsToConst(DataPoint* centroids) {
    Data_T values[KSize*FeatSize];
    
    for(int i=0; i!=KSize; ++i) {
        for(int j=0; j!=FeatSize; ++j) {
            values[i*FeatSize+j] = centroids[i].value[j];
        }
    }
    cudaAssert (
        hipMemcpyToSymbol(HIP_SYMBOL(constCentroidValues), values, KSize*FeatSize*sizeof(Data_T))
    );
}

void memcpyCentroidsFromConst(DataPoint* centroids) {
    Label_T labels[KSize];
    Data_T values[KSize*FeatSize];

    cudaAssert (
        hipMemcpyFromSymbol(values, HIP_SYMBOL(constCentroidValues), KSize*FeatSize*sizeof(Data_T))
    );

    for(int i=0; i!=KSize; ++i) {
        centroids[i].label = labels[i];

        for(int j=0; j!=FeatSize; ++j) {
            centroids[i].value[j] = values[i*FeatSize+j];
        }
    }
}

void memcpyLabelCountToConst(size_t* labelCount, size_t* labelFirstIdxes, size_t* labelLastIdxes) {
    cudaAssert (
        hipMemcpyToSymbol(HIP_SYMBOL(constLabelCounts), labelCount, KSize*sizeof(size_t))
    );
    cudaAssert (
        hipMemcpyToSymbol(HIP_SYMBOL(constLabelFirstIdxes), labelFirstIdxes, KSize*sizeof(size_t))
    );
    cudaAssert (
        hipMemcpyToSymbol(HIP_SYMBOL(constLabelLastIdxes), labelLastIdxes, KSize*sizeof(size_t))
    );
}

void memcpyLabelCountFromConst(size_t* labelCount, size_t* labelFirstIdxes, size_t* labelLastIdxes) {
    cudaAssert (
        hipMemcpyFromSymbol(labelCount, HIP_SYMBOL(constLabelCounts), KSize*sizeof(size_t))
    );
    cudaAssert (
        hipMemcpyFromSymbol(labelFirstIdxes, HIP_SYMBOL(constLabelFirstIdxes), KSize*sizeof(size_t))
    );
    cudaAssert (
        hipMemcpyFromSymbol(labelLastIdxes, HIP_SYMBOL(constLabelLastIdxes), KSize*sizeof(size_t))
    );
}
