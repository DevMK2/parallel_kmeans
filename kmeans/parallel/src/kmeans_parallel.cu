#include "hip/hip_runtime.h"
#include "kmeans_parallel.cuh"
#include "announce.hh"

void KMeans::main(DataPoint* const centroids, DataPoint* const data) {
    cudaAssert (
        hipHostRegister(data, DataSize*sizeof(DataPoint), hipHostRegisterPortable)
    );
    cudaAssert (
        hipHostRegister(centroids, KSize*sizeof(DataPoint), hipHostRegisterPortable)
    );

    auto newCentroids = new DataPoint[KSize];
    bool* isSame = new bool(true);

    cudaAssert (
        hipHostRegister(newCentroids, KSize*sizeof(DataPoint), hipHostRegisterPortable)
    );
    cudaAssert (
        hipHostRegister(isSame, sizeof(bool), hipHostRegisterPortable)
    );

    //study(deviceQuery());
    int numThread_labeling = 128; /*TODO get from study*/
    int numBlock_labeling = ceil((float)DataSize / numThread_labeling);

    int threashold = 3; // 
    while(threashold-- > 0) {
        KMeans::labeling<<<numBlock_labeling, numThread_labeling>>>(centroids, data);
        //hipDeviceSynchronize();
        //announce.Labels(data);

        resetNewCentroids<<<KSize,FeatSize>>>(newCentroids);

        KMeans::updateCentroidAccum<<<numBlock_labeling,numThread_labeling>>>(newCentroids, data);
        KMeans::updateCentroidDivide<<<KSize, FeatSize>>>(newCentroids);

        KMeans::checkIsSame<<<KSize, FeatSize>>>(isSame, centroids, newCentroids);
        //hipDeviceSynchronize();
        //if(isSame)
            //break;

        memcpyCentroid<<<KSize,FeatSize>>>(centroids, newCentroids);
    }
    hipDeviceSynchronize();
    cudaAssert( hipPeekAtLastError());
    announce.Labels(data);

    cudaAssert( hipHostUnregister(data) );
    cudaAssert( hipHostUnregister(centroids) );
    cudaAssert( hipHostUnregister(newCentroids) );
    cudaAssert( hipHostUnregister(isSame) );

    delete[] newCentroids;
    delete isSame;
}

/// labeling ////////////////////////////////////////////////////////////////////////////////////
__global__
void KMeans::labeling(const DataPoint* const centroids, DataPoint* const data) {
    const int& idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= DataSize)
        return;

    const DataPoint* centroidPtr = centroids;
    DataPoint threadData = data[idx];

    Label_T minDistLabel = 0;
    Data_T minDistSQR = MaxDataValue;

    for(int i=0; i!=KSize; ++i) {
        Data_T currDistSQR = Labeling::euclideanDistSQR(threadData.value, centroidPtr->value);
        if(minDistSQR > currDistSQR) {
            minDistLabel = i;
            minDistSQR = currDistSQR;
        }

        centroidPtr++;
    }

    data[idx].label = minDistLabel;
}

__device__
Data_T KMeans::Labeling::euclideanDistSQR (const Data_T* const lhs, const Data_T* const rhs) {
    const Data_T* valuePtrLHS = lhs;
    const Data_T* valuePtrRHS = rhs;

    Data_T distSQR = 0;

    for(int featIdx=0; featIdx!=FeatSize; ++featIdx) {
        Data_T dist = *valuePtrLHS - *valuePtrRHS;

        distSQR += dist*dist;

        valuePtrLHS++;
        valuePtrRHS++;
    }

    return distSQR;
}

/// update centroids //////////////////////////////////////////////////////////////////////////////
__global__
void KMeans::updateCentroidAccum(DataPoint* const centroids, const DataPoint* const data) {
    const int dataIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if(dataIdx >= DataSize)
        return;

    const int centroidIdx = data[dataIdx].label;

    atomicAdd(&(centroids[centroidIdx].label), 1); // newCentroids는 labelSize를 나타내기 위해 0으로 초기화됨
    Update::addValuesLtoR(data[dataIdx].value, centroids[centroidIdx].value);
}

__global__
void KMeans::updateCentroidDivide(DataPoint* const centroids) {
    centroids[blockIdx.x].value[threadIdx.x] /= centroids[blockIdx.x].label;
}

__device__
void KMeans::Update::addValuesLtoR(const Data_T* const lhs, Data_T* const rhs) {
    const Data_T* lhsPtr = lhs;
    Data_T* rhsPtr = rhs;

    for(int featIdx=0; featIdx!=FeatSize; ++featIdx)
        atomicAdd(rhsPtr++, *(lhsPtr++));
}

void study(const std::vector<DeviceQuery>& devices) {
    /*
     * According to the CUDA C Best Practice Guide.
     * 1. Thread per block should be a multiple of 32(warp size)
     * 2. A minimum of 64 threads per block should be used.
     * 3. Between 128 and 256 thread per block is a better choice
     * 4. Use several(3 to 4) small thread blocks rather than one large thread block
     */
    /* 
     * sizeof DataPoint 
     *   = 4(float) * 200(feature size) + 4(label, int) 
     *   = 804 byte
     *   =>register memory per thread
     *     = 832 byte { 804 + 8(pointer) + 8(two int) + 8(size_t) + 4(Data_T) }
     *   =>register count per thread
     *     = 832/4 = 208
     *
     * sizeof Centroid
     *   = DataPoint x 10
     *   = 8040 byte
     * 
     * memory per block (* NOT SHARED MEMORY *)
     *   = 804 * 64 
     *   = 51456 byte
     *
     * total global memory size = 8112 MBytes
     * number of registers per block = 65536
     */
    Count_T numRegisterPerKernel_labeling = 208;
    MemSize_L sizeDataPoint = sizeof(DataPoint);
    MemSize_L sizeCentroids = sizeDataPoint * KSize;
    for(auto device : devices) {
        assert(sizeCentroids < device.totalConstMem);

        std::cout <<  "Device["<<device.index<<"]" << std::endl;

        Count_T maxThreadsPerBlock = device.numRegPerBlock / numRegisterPerKernel_labeling;
        std::cout <<"max threads per block(labeling) : " << maxThreadsPerBlock << std::endl;
        std::cout <<"max threads per block(update)   : " << maxThreadsPerBlock << std::endl;
        std::cout <<"max threads per block(check)    : " << maxThreadsPerBlock << std::endl;

        std::cout << device.numRegPerBlock / 208.0 << std::endl;
        std::cout << device.threadsPerBlock << std::endl;
        std::cout << device.threadsPerMultiprocesser << std::endl;
    }
}