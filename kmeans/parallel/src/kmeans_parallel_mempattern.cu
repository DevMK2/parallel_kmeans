#include "hip/hip_runtime.h"
#include "kmeans_parallel.cuh"
#include "announce.hh"
#include <algorithm>
#include "log.cc"

static const int UpdateCentroidBlockDim = 1024;

void memcpyCentroidsToConst(DataPoint*);
void memcpyCentroidsFromConst(DataPoint*);
void memcpyLabelCountToConst(size_t*, size_t*, size_t*);
void memcpyLabelCountFromConst(size_t*, size_t*, size_t*);

__device__ __constant__ Data_T constCentroidValues[KSize*FeatSize];
__device__ __constant__ size_t constLabelCounts[KSize];
__device__ __constant__ size_t constLabelFirstIdxes[KSize];
__device__ __constant__ size_t constLabelLastIdxes[KSize];

#define Trans_DataValues_IDX(x,y) y*DataSize+x
#define CentroidValues_IDX(x,y) y*FeatSize+x
__global__
void transposeDataPointers(const DataPoint* const data, Labels_T labels, Trans_DataValues transposed);
__global__
void untransposeDataPointers(const Trans_DataValues transposed, Labels_T labels, DataPoint* const data);

void calcLabelCounts (
    Labels_T const dataLabels,
    Label_T* const dataIDXs,
    size_t* const labelCounts
);

void setLabelBounds (
    const size_t* const labelCounts,
    size_t* const labelFirstIdxes,
    size_t* const labelLastIdxes
);

__global__
void sortDatapoints (
    Labels_T const dataLabels,
    const Label_T* const dataIDXs,
    Trans_DataValues const dataValuesTransposed,
    Trans_DataValues const newDataValuesTransposed
);

void KMeans::main(DataPoint* const centroids, DataPoint* const data) {
#ifdef SPARSE_LOG
    Log<> log ( 
        LogFileName.empty()?  "./results/parallel_mempattern" : LogFileName
    );
#endif

    Labels_T dataLabels = new Label_T[DataSize];
    Trans_DataValues dataValuesTransposed = new Data_T[FeatSize * DataSize];
    Trans_DataValues newDataValuesTransposed;
    cudaAssert (
        hipHostRegister(dataValuesTransposed, FeatSize*DataSize*sizeof(Data_T), hipHostRegisterPortable)
    );
    cudaAssert (
        hipMalloc((void**)&newDataValuesTransposed, FeatSize*DataSize*sizeof(Data_T))
    );
    cudaAssert (
        hipHostRegister(data, DataSize*sizeof(DataPoint), hipHostRegisterPortable)
    );
    cudaAssert (
        hipHostRegister(dataLabels, DataSize*sizeof(Label_T), hipHostRegisterPortable)
    );
    cudaAssert (
        hipHostRegister(centroids, KSize*sizeof(DataPoint), hipHostRegisterPortable)
    );
    transposeDataPointers<<<DataSize, FeatSize>>>(data, dataLabels, dataValuesTransposed);

    auto dataIDXs = new int[DataSize]{0,};
    auto labelCounts = new size_t[KSize]{0,};
    auto labelFirstIdxes = new size_t[KSize]{0,};
    auto labelLastIdxes = new size_t[KSize]{0,};
    auto newCentroids = new DataPoint[KSize];
    auto isUpdated = new bool(true);

    cudaAssert (
        hipHostRegister(dataIDXs, DataSize*sizeof(int), hipHostRegisterPortable)
    );
    cudaAssert (
        hipHostRegister(newCentroids, KSize*sizeof(DataPoint), hipHostRegisterPortable)
    );
    cudaAssert (
        hipHostRegister(isUpdated, sizeof(bool), hipHostRegisterPortable)
    );

    int numThread_labeling = 256; /*TODO calc from study*/
    int numBlock_labeling = ceil((float)DataSize / numThread_labeling);

#ifdef DEEP_LOG
    Log<LoopEvaluate, 1024> deeplog (
        LogFileName.empty()?  "./results/parallel_mampattern_deep" : LogFileName+"_deep"
    );
#endif
    while(threashold--) {
        hipDeviceSynchronize();
        memcpyCentroidsToConst(centroids);
        KMeans::labeling<<<numBlock_labeling, numThread_labeling>>>(dataLabels, dataValuesTransposed);
hipDeviceSynchronize();
cudaAssert( hipPeekAtLastError());

        hipDeviceSynchronize();
#ifdef DEEP_LOG
        deeplog.Lap("labeling");
        untransposeDataPointers<<<DataSize, FeatSize>>>(dataValuesTransposed, dataLabels, data);
        hipDeviceSynchronize();
        announce.Labels(data);
#endif
///////////////////////////////////////////////////////////////
        calcLabelCounts(dataLabels, dataIDXs, labelCounts);
#ifdef DEEP_LOG
        deeplog.Lap("sorting1");
#endif
        setLabelBounds(labelCounts, labelFirstIdxes, labelLastIdxes);
#ifdef DEEP_LOG
        deeplog.Lap("sorting2");
#endif
        memcpyLabelCountToConst(labelCounts, labelFirstIdxes, labelLastIdxes);
#ifdef DEEP_LOG
        deeplog.Lap("sorting3");
#endif
        sortDatapoints<<<numBlock_labeling, numThread_labeling>>> (
            dataLabels, dataIDXs, dataValuesTransposed, newDataValuesTransposed
        );
        hipMemcpy(dataValuesTransposed, newDataValuesTransposed, DataSize*FeatSize*sizeof(Data_T), hipMemcpyDeviceToDevice);
hipDeviceSynchronize();
cudaAssert( hipPeekAtLastError());
#ifdef DEEP_LOG
        deeplog.Lap("sorting4");
#endif
///////////////////////////////////////////////////////////////
        size_t maxLabelCount = 0;
        for(int i=0; i!=KSize; ++i)
            maxLabelCount = std::max(maxLabelCount, labelCounts[i]);

        resetNewCentroids<<<KSize,FeatSize>>>(newCentroids);
hipDeviceSynchronize();
cudaAssert( hipPeekAtLastError());
        dim3 dimBlock(UpdateCentroidBlockDim, 1, 1);
        dim3 dimGrid(ceil((float)maxLabelCount/UpdateCentroidBlockDim), KSize, 1);
        KMeans::updateCentroidAccum<<<dimGrid, dimBlock>>>(newCentroids, dataValuesTransposed);
hipDeviceSynchronize();
cudaAssert( hipPeekAtLastError());
        KMeans::updateCentroidDivide<<<KSize, FeatSize>>>(newCentroids);
hipDeviceSynchronize();
cudaAssert( hipPeekAtLastError());
#ifdef DEEP_LOG
        hipDeviceSynchronize();
        deeplog.Lap("updateCentroid");
#endif

        KMeans::checkIsSame<<<KSize, FeatSize>>>(isUpdated, centroids, newCentroids);
        hipDeviceSynchronize();
        if(*isUpdated)
            break;
        *isUpdated = false;

        memcpyCentroid<<<KSize,FeatSize>>>(centroids, newCentroids);
#ifdef DEEP_LOG
        deeplog.Lap("check centroids");
#endif
    }

    hipDeviceSynchronize();
    cudaAssert( hipPeekAtLastError());
#ifdef SPARSE_LOG
    log.Lap("KMeans-Parallel-MemPattern");
#endif
    announce.Labels(data);
    announce.InitCentroids(newCentroids);

    cudaAssert( hipHostUnregister(data));
    cudaAssert( hipHostUnregister(dataIDXs));
    cudaAssert( hipHostUnregister(dataLabels));
    cudaAssert( hipHostUnregister(dataValuesTransposed));
    cudaAssert( hipHostUnregister(centroids) );
    cudaAssert( hipHostUnregister(newCentroids) );
    cudaAssert( hipHostUnregister(isUpdated) );
    cudaAssert( hipFree(newDataValuesTransposed));

    delete[] dataIDXs;
    delete[] dataLabels;
    delete[] dataValuesTransposed;
    delete[] labelCounts;
    delete[] labelFirstIdxes;
    delete[] labelLastIdxes;
    delete[] newCentroids;
    delete isUpdated;
}

__global__
void KMeans::labeling(Labels_T const labels, Trans_DataValues const data) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= DataSize)
        return;

    Data_T distSQRSums[KSize]{0,};

    for(int i=0; i!=FeatSize; ++i) {
        Data_T currValue = data[i*DataSize+ idx];

        for(int j=0; j!=KSize; ++j) {
            Data_T currDist = currValue - constCentroidValues[j*FeatSize + i];
            distSQRSums[j] += currDist * currDist;
        }
        __syncthreads();
    }

    Data_T minDistSQRSum = MaxDataValue;
    Label_T minDistLabel = 0;
    for(int i=0; i!=KSize; ++i) {
        if(minDistSQRSum > distSQRSums[i]) {
            minDistSQRSum = distSQRSums[i];
            minDistLabel = i;
        }
    }

    labels[idx] = minDistLabel;
}

/// update centroids //////////////////////////////////////////////////////////////////////////////
// blockDim = 8~32 ~ 128
// gridDim = 10, ceil(maxLabelCount/blockDim)
// width = maxLabelCount
// blockIdx.y = label
__global__
void KMeans::updateCentroidAccum(DataPoint* const centroids, const Trans_DataValues data) {
    __shared__ Data_T Sum[UpdateCentroidBlockDim];

    const int tID = threadIdx.x;
    const Label_T label = blockIdx.y;

    const size_t labelFirstIdx = constLabelFirstIdxes[label]; // TODO Delete me
    const size_t labelLastIdx = constLabelLastIdxes[label];   // TODO Delete me
    const size_t dataIdx = labelFirstIdx + (blockIdx.x * blockDim.x + tID);

    if(dataIdx > labelLastIdx)
        return;

    {//\Asserts
    assert(label >= 0 && label < 10);
    assert(labelLastIdx < DataSize);
    assert(dataIdx >= labelFirstIdx);
    }

    for(int featIdx=0; featIdx!=FeatSize; ++featIdx) {
        Sum[tID] = data[Trans_DataValues_IDX(dataIdx, featIdx)];
        __syncthreads();// TODO 없어도 되나?

        for(int stride=blockDim.x/2; stride>=1; stride>>=1) {
            if(tID < stride && dataIdx+stride <= labelLastIdx)
                Sum[tID] += Sum[tID+stride];
            __syncthreads();
        }

        if(tID != 0)
            continue;

        atomicAdd(&(centroids[label].value[featIdx]), Sum[tID]);
    }
}

__global__
void KMeans::updateCentroidDivide(DataPoint* const centroids) {
    int label = blockIdx.x;
    centroids[label].value[threadIdx.x] /= constLabelCounts[label];
}

void memcpyCentroidsToConst(DataPoint* centroids) {
    Data_T values[KSize*FeatSize];
    
    for(int i=0; i!=KSize; ++i) {
        for(int j=0; j!=FeatSize; ++j) {
            values[i*FeatSize+j] = centroids[i].value[j];
        }
    }
    cudaAssert (
        hipMemcpyToSymbol(HIP_SYMBOL(constCentroidValues), values, KSize*FeatSize*sizeof(Data_T))
    );
}

void memcpyCentroidsFromConst(DataPoint* centroids) {
    Label_T labels[KSize];
    Data_T values[KSize*FeatSize];

    cudaAssert (
        hipMemcpyFromSymbol(values, HIP_SYMBOL(constCentroidValues), KSize*FeatSize*sizeof(Data_T))
    );

    for(int i=0; i!=KSize; ++i) {
        centroids[i].label = labels[i];

        for(int j=0; j!=FeatSize; ++j) {
            centroids[i].value[j] = values[i*FeatSize+j];
        }
    }
}

void memcpyLabelCountToConst(size_t* labelCount, size_t* labelFirstIdxes, size_t* labelLastIdxes) {
    cudaAssert (
        hipMemcpyToSymbol(HIP_SYMBOL(constLabelCounts), labelCount, KSize*sizeof(size_t))
    );
    cudaAssert (
        hipMemcpyToSymbol(HIP_SYMBOL(constLabelFirstIdxes), labelFirstIdxes, KSize*sizeof(size_t))
    );
    cudaAssert (
        hipMemcpyToSymbol(HIP_SYMBOL(constLabelLastIdxes), labelLastIdxes, KSize*sizeof(size_t))
    );
}

void memcpyLabelCountFromConst(size_t* labelCount, size_t* labelFirstIdxes, size_t* labelLastIdxes) {
    cudaAssert (
        hipMemcpyFromSymbol(labelCount, HIP_SYMBOL(constLabelCounts), KSize*sizeof(size_t))
    );
    cudaAssert (
        hipMemcpyFromSymbol(labelFirstIdxes, HIP_SYMBOL(constLabelFirstIdxes), KSize*sizeof(size_t))
    );
    cudaAssert (
        hipMemcpyFromSymbol(labelLastIdxes, HIP_SYMBOL(constLabelLastIdxes), KSize*sizeof(size_t))
    );
}

__global__
void transposeDataPointers(const DataPoint* const data, Labels_T labels, Trans_DataValues transposed) {
    int dataIdx = blockIdx.x;
    int valueIdx = threadIdx.x;

    if(valueIdx==0)
        labels[dataIdx] = data[dataIdx].label;

    transposed[valueIdx*DataSize + dataIdx] = data[dataIdx].value[valueIdx];
}

__global__
void untransposeDataPointers(const Trans_DataValues transposed, Labels_T labels, DataPoint* const data) {
    int dataIdx = blockIdx.x;
    int valueIdx = threadIdx.x;

    if(valueIdx==0)
        data[dataIdx].label = labels[dataIdx];

    data[dataIdx].value[valueIdx] = transposed[valueIdx*DataSize + dataIdx];
}

void calcLabelCounts (
    Labels_T const dataLabels,
    Label_T* const dataIDXs,
    size_t* const labelCounts
) {
    memset(labelCounts, 0, KSize*sizeof(size_t));

    for(int i=0; i!=DataSize; ++i) {
        Label_T curr = dataLabels[i];
        dataIDXs[i] = labelCounts[curr];// 라벨 중에 몇 번째인지 index
        labelCounts[curr] += 1; // 라벨 당 datapoint 갯수
    }
}

void setLabelBounds (
    const size_t* const labelCounts,
    size_t* const labelFirstIdxes,
    size_t* const labelLastIdxes
) {
    labelFirstIdxes[0] = 0;
    labelLastIdxes[0] = labelCounts[0] - 1;

    for(int i=1; i!=KSize; ++i) {
        labelFirstIdxes[i] = labelLastIdxes[i-1] + 1;
        labelLastIdxes[i] = labelFirstIdxes[i] + labelCounts[i] - 1;
    }
}

__global__
void sortDatapoints (
    Labels_T const dataLabels,
    const Label_T* const dataIDXs,
    Trans_DataValues const dataValuesTransposed,
    Trans_DataValues const newDataValuesTransposed
) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= DataSize)
        return;

    int targetIDX = constLabelFirstIdxes[dataLabels[idx]] + dataIDXs[idx];

    for(int j=0; j!=FeatSize; ++j) {
        int row = j*DataSize;
        newDataValuesTransposed[row+targetIDX] = dataValuesTransposed[row+idx];
    }
}