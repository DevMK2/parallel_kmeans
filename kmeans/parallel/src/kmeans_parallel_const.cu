#include "hip/hip_runtime.h"
#include "kmeans_parallel.cuh"
#include "announce.hh"
#include "log.cc"

void memcpyCentroidsToConst(DataPoint* centroids);
void memcpyCentroidsFromConst(DataPoint* centroids);

__device__ __constant__ Data_T constCentroidValues[KSize*FeatSize];

void KMeans::main(DataPoint* const centroids, DataPoint* const data) {
#ifdef SPARSE_LOG
    Log<> log ( 
        LogFileName.empty()?  "./results/parallel_const" : LogFileName
    );
#endif
    cudaAssert (
        hipHostRegister(data, DataSize*sizeof(DataPoint), hipHostRegisterPortable)
    );
    cudaAssert (
        hipHostRegister(centroids, KSize*sizeof(DataPoint), hipHostRegisterPortable)
    );

    auto newCentroids = new DataPoint[KSize];
    bool* isUpdated = new bool(true);

    cudaAssert (
        hipHostRegister(newCentroids, KSize*sizeof(DataPoint), hipHostRegisterPortable)
    );
    cudaAssert (
        hipHostRegister(isUpdated, sizeof(bool), hipHostRegisterPortable)
    );

    //study(deviceQuery());
    int numThread_labeling = 8; /*TODO get from study*/
    int numBlock_labeling = ceil((float)DataSize / numThread_labeling);

#ifdef DEEP_LOG
    Log<LoopEvaluate, 1024> deeplog (
        LogFileName.empty()?  "./results/parallel_const_deep" : LogFileName+"_deep"
    );
#endif
    while(threashold--) {
        hipDeviceSynchronize();
        memcpyCentroidsToConst(centroids);
        KMeans::labeling<<<numBlock_labeling, numThread_labeling>>>(data);
#ifdef DEEP_LOG
        hipDeviceSynchronize();
        announce.Labels(data);
        deeplog.Lap("labeling");
#endif

        resetNewCentroids<<<KSize,FeatSize>>>(newCentroids);

        KMeans::updateCentroidAccum<<<numBlock_labeling,numThread_labeling>>>(newCentroids, data);
        KMeans::updateCentroidDivide<<<KSize, FeatSize>>>(newCentroids);
#ifdef DEEP_LOG
        hipDeviceSynchronize();
        deeplog.Lap("updateCentroid");
#endif

        KMeans::checkIsSame<<<KSize, FeatSize>>>(isUpdated, centroids, newCentroids);
        hipDeviceSynchronize();
        if(*isUpdated)
            break;
        *isUpdated = false;
        memcpyCentroid<<<KSize,FeatSize>>>(centroids, newCentroids);
#ifdef DEEP_LOG
        deeplog.Lap("check centroids");
#endif
    }
    hipDeviceSynchronize();
    cudaAssert( hipPeekAtLastError());
#ifdef SPARSE_LOG
    log.Lap("KMeans-Parallel-const End");
#endif
    announce.Labels(data);
    announce.InitCentroids(newCentroids);

    cudaAssert( hipHostUnregister(data) );
    cudaAssert( hipHostUnregister(centroids) );
    cudaAssert( hipHostUnregister(newCentroids) );
    cudaAssert( hipHostUnregister(isUpdated) );

    delete[] newCentroids;
    delete isUpdated;
}

/// labeling ////////////////////////////////////////////////////////////////////////////////////
__global__
void KMeans::labeling(DataPoint* const data) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= DataSize)
        return;

    DataPoint threadData = data[idx];

    Label_T minDistLabel = 0;
    Data_T minDistSQR = MaxDataValue;

    for(int i=0; i!=KSize; ++i) {
        Data_T currDistSQR = KMeans::Labeling::euclideanDistSQR(threadData.value, constCentroidValues + i*FeatSize);
        if(minDistSQR > currDistSQR) {
            minDistLabel = i;
            minDistSQR = currDistSQR;
        }
    }

    data[idx].label = minDistLabel;
}

__device__ 
Data_T KMeans::Labeling::euclideanDistSQR ( const Data_T* const __restrict__ lhs, const Data_T* const __restrict__ rhs) { 
    const Data_T* valuePtrLHS = lhs;
    const Data_T* valuePtrRHS = rhs;

    Data_T distSQR = 0;

    for(int i=0; i!=FeatSize; ++i) {
        Data_T dist = *valuePtrLHS - *valuePtrRHS;

        distSQR += dist*dist;

        valuePtrLHS++;
        valuePtrRHS++;
    }

    return distSQR;
}

/// update centroids //////////////////////////////////////////////////////////////////////////////
__global__
void KMeans::updateCentroidAccum(DataPoint* const centroids, const DataPoint* const data) {
    const int dataIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if(dataIdx >= DataSize)
        return;

    const int centroidIdx = data[dataIdx].label;

    atomicAdd(&(centroids[centroidIdx].label), 1); // newCentroids는 labelSize를 나타내기 위해 0으로 초기화됨
    Update::addValuesLtoR(data[dataIdx].value, centroids[centroidIdx].value);
}

__global__
void KMeans::updateCentroidDivide(DataPoint* const centroids) {
    centroids[blockIdx.x].value[threadIdx.x] /= centroids[blockIdx.x].label;
}

__device__
void KMeans::Update::addValuesLtoR(const Data_T* const lhs, Data_T* const rhs) {
    const Data_T* lhsPtr = lhs;
    Data_T* rhsPtr = rhs;

    for(int featIdx=0; featIdx!=FeatSize; ++featIdx)
        atomicAdd(rhsPtr++, *(lhsPtr++));
}

void memcpyCentroidsFromConst(DataPoint* centroids) {
    Label_T labels[KSize];
    Data_T values[KSize*FeatSize];

    cudaAssert (
        hipMemcpyFromSymbol(values, HIP_SYMBOL(constCentroidValues), KSize*FeatSize*sizeof(Data_T))
    );

    for(int i=0; i!=KSize; ++i) {
        centroids[i].label = labels[i];

        for(int j=0; j!=FeatSize; ++j) {
            centroids[i].value[j] = values[i*FeatSize+j];
        }
    }
}

void memcpyCentroidsToConst(DataPoint* centroids) {
    Data_T values[KSize*FeatSize];
    
    for(int i=0; i!=KSize; ++i) {
        for(int j=0; j!=FeatSize; ++j) {
            values[i*FeatSize+j] = centroids[i].value[j];
        }
    }
    cudaAssert (
        hipMemcpyToSymbol(HIP_SYMBOL(constCentroidValues), values, KSize*FeatSize*sizeof(Data_T))
    );
}
